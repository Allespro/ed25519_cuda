#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <memory.h>
#include <assert.h>

unsigned int PERF_BLOCKS = 1024;
unsigned int PERF_THREADS = 256;

#include "ed25519.cuh"
#include "ge.cuh"
#include "sc.cuh"

void pubkey(unsigned char *seed, unsigned char *pk)
{
  uint8_t h[64];
  ge_p3 A;
  sha512(seed,32,h);
  h[0] &= 248;
  h[31] &= 63;
  h[31] |= 64;
  //ge_scalarmult_base(&A,h);
  //ge_p3_tobytes(pk,&A);

}

void display_details(unsigned char *public_key_h, unsigned char *private_key_h) {
    printf("Public Key\n");
    for (int i = 0; i < 32; ++i) {
        printf("%d  ", public_key_h[i]);
    }
    printf("\n");

    printf("Private Key\n");
    for (int i = 0; i < 64; ++i) {
        printf("%d  ", private_key_h[i]);
    }
    printf("\n");
    onion_address(public_key_h);
}

void display_key_pair(unsigned char public_keys[][33], unsigned char private_keys[][65], int n) {
    printf("Logging the key pairs after scalar addition of \"aaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaa\" :\n\n");

    for (int j = 0; j < n; ++j) {
        printf("Public Key %d\n", j + 1);
        for (int i = 0; i < 32; ++i) {
            printf("%d  ", public_keys[j][i]);
        }
        printf("\n");

        printf("Private Key %d\n", j + 1);
        for (int i = 0; i < 64; ++i) {
            printf("%d  ", private_keys[j][i]);
        }
        printf("\n\n");
    }
}

void display_secret(unsigned char secret[2][33]) {
    printf("Following are the secrets for the key exchange (both secret should be same):\n\n");

    for(int j = 0; j < 2; ++j) {
        printf("Secret %d:\n", j + 1);
        for (int i = 0; i < 32; ++i) {
            printf("%d  ", secret[j][i]);
        }
        printf("\n");
    }

}

int create_keypair(bool enable_logging, bool test_seed) {
    unsigned char *public_key;
    unsigned char *private_key;
    unsigned char *seed_hf;
    unsigned char *seed;
    unsigned char seed_h[33] = "01234567890123456789012345678901";

    hipMalloc(&public_key, 32 * sizeof(unsigned char));
    hipMalloc(&private_key, 64 * sizeof(unsigned char));
    hipMalloc(&seed, 32 * sizeof(unsigned char));
    hipHostMalloc(&seed_hf, 32 * sizeof(unsigned char));

    // Test whether create seed __host__ function works
    ed25519_kernel_create_seed(seed_hf,1);
    hipMemcpy(seed, seed_hf, 32 * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (test_seed) {
        hipMemcpy(seed, seed_h, 32 * sizeof(unsigned char), hipMemcpyHostToDevice);
    }

    ed25519_kernel_create_keypair_batch<<<1,1>>>(public_key, private_key, (const unsigned char*) seed, 1);

    if (enable_logging) {
        unsigned char public_key_h[32];
        unsigned char private_key_h[64];
        hipMemcpy(public_key_h, public_key, 32 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        hipMemcpy(private_key_h, private_key, 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        display_details(public_key_h, private_key_h);
    }

    hipFree(public_key);
    hipFree(private_key);
    hipFree(seed);
    hipHostFree(seed_hf);
    return 1;
}

int main(int argc, char **argv) {
    bool enable_logging = false;
    bool test_seed = false;
    for (int i = 0 ; i < argc; ++i) {
        if (strcmp(argv[i], "--logging") == 0 || strcmp(argv[i], "-l") == 0) {
            enable_logging = true;
        } else if (strcmp(argv[i], "--testseed") == 0 || strcmp(argv[i], "-t") == 0) {
            test_seed = true;
        }
    }

    printf(((enable_logging)? "\n\n---------- Test batch with single key pair ----------\n\n": ""));
    assert(create_keypair(enable_logging, test_seed));
    return 0;
}
