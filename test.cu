#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <memory.h>
#include <assert.h>

unsigned int PERF_BLOCKS = 1024;
unsigned int PERF_THREADS = 256;

#include "ed25519.cuh"
#include "ge.cuh"
#include "sc.cuh"
#include "onion.cuh"

void display_details(unsigned char *public_key_h, unsigned char *private_key_h, uint8_t *checksum_h) {
    
    printf("Public Key\n");
    for (int i = 0; i < 32; ++i) {
        printf("%d  ", public_key_h[i]);
    }
    printf("\n");

    printf("Private Key\n");
    for (int i = 0; i < 64; ++i) {
        printf("%d  ", private_key_h[i]);
    }
    printf("\n");

    
    printf("Checksum\n");
    for (int i = 0; i < 64; ++i) {
        printf("%d  ", checksum[i]);
    }
    printf("\n");
}

void display_key_pair(unsigned char public_keys[][33], unsigned char private_keys[][65], int n) {
    printf("Logging the key pairs after scalar addition of \"aaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaa\" :\n\n");

    for (int j = 0; j < n; ++j) {
        printf("Public Key %d\n", j + 1);
        for (int i = 0; i < 32; ++i) {
            printf("%d  ", public_keys[j][i]);
        }
        printf("\n");

        printf("Private Key %d\n", j + 1);
        for (int i = 0; i < 64; ++i) {
            printf("%d  ", private_keys[j][i]);
        }
        printf("\n\n");
    }
}

void display_secret(unsigned char secret[2][33]) {
    printf("Following are the secrets for the key exchange (both secret should be same):\n\n");

    for(int j = 0; j < 2; ++j) {
        printf("Secret %d:\n", j + 1);
        for (int i = 0; i < 32; ++i) {
            printf("%d  ", secret[j][i]);
        }
        printf("\n");
    }

}

int create_keypair(bool enable_logging, bool test_seed) {
    unsigned char *public_key;
    unsigned char *private_key;
    unsigned char *seed_hf;
    unsigned char *seed;
    uint8_t *checksum;
    unsigned char seed_h[33] = "01234567890123456789012345678901";
    uint8_t checksum_h[200] = ".onion checksum";

    hipMalloc(&public_key, 32 * sizeof(unsigned char));
    hipMalloc(&private_key, 64 * sizeof(unsigned char));
    hipMalloc(&seed, 32 * sizeof(unsigned char));
    hipHostMalloc(&seed_hf, 32 * sizeof(unsigned char));

    // Test whether create seed __host__ function works
    ed25519_kernel_create_seed(seed_hf,1);
    hipMemcpy(seed, seed_hf, 32 * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (test_seed) {
        hipMemcpy(seed, seed_h, 32 * sizeof(unsigned char), hipMemcpyHostToDevice);
    }
    ed25519_kernel_create_keypair_batch<<<1,1>>>(public_key, private_key, (const unsigned char*) seed, 1);

    hipMemcpy(checksum, checksum_h, 200 * sizeof(uint8_t), hipMemcpyHostToDevice);
    onion_address<<<1,1>>>(public_key, (uint8_t*) checksum);

    if (enable_logging) {
        unsigned char public_key_h[32];
        unsigned char private_key_h[64];
        hipMemcpy(public_key_h, public_key, 32 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        hipMemcpy(private_key_h, private_key, 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        hipMemcpy(checksum_h, checksum, 200 * sizeof(uint8_t), hipMemcpyDeviceToHost);
        display_details(public_key_h, private_key_h, checksum_h);
    }

    hipFree(public_key);
    hipFree(private_key);
    hipFree(seed);
    hipHostFree(seed_hf);
    return 1;
}

int main(int argc, char **argv) {
    bool enable_logging = false;
    bool test_seed = false;
    for (int i = 0 ; i < argc; ++i) {
        if (strcmp(argv[i], "--logging") == 0 || strcmp(argv[i], "-l") == 0) {
            enable_logging = true;
        } else if (strcmp(argv[i], "--testseed") == 0 || strcmp(argv[i], "-t") == 0) {
            test_seed = true;
        }
    }

    printf(((enable_logging)? "\n\n---------- Test batch with single key pair ----------\n\n": ""));
    assert(create_keypair(enable_logging, test_seed));
    return 0;
}
