#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <memory.h>
#include <assert.h>

unsigned int PERF_BLOCKS = 1024;
unsigned int PERF_THREADS = 256;

#include "src/ed25519.cuh"
#include "src/ge.cuh"
#include "src/sc.cuh"


void display_details(unsigned char *public_key_h, unsigned char *private_key_h) {
    printf("Logging the first public key and private key of the batch along with the signature for the input:\n\"9d61b19deffd5a60ba844af492ec2cc44449c5697b326919703bac031cae7f60d75a980182b10ab7d54bfed3c964073a0ee172f3daa62325af021a68f707511a\" :\n\n");

    printf("Public Key\n");
    for (int i = 0; i < 32; ++i) {
        printf("%d  ", public_key_h[i]);
    }
    printf("\n");

    printf("Private Key\n");
    for (int i = 0; i < 64; ++i) {
        printf("%d  ", private_key_h[i]);
    }
    printf("\n");
}

void display_key_pair(unsigned char public_keys[][33], unsigned char private_keys[][65], int n) {
    printf("Logging the key pairs after scalar addition of \"aaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaa\" :\n\n");

    for (int j = 0; j < n; ++j) {
        printf("Public Key %d\n", j + 1);
        for (int i = 0; i < 32; ++i) {
            printf("%d  ", public_keys[j][i]);
        }
        printf("\n");

        printf("Private Key %d\n", j + 1);
        for (int i = 0; i < 64; ++i) {
            printf("%d  ", private_keys[j][i]);
        }
        printf("\n\n");
    }
}

void display_secret(unsigned char secret[2][33]) {
    printf("Following are the secrets for the key exchange (both secret should be same):\n\n");

    for(int j = 0; j < 2; ++j) {
        printf("Secret %d:\n", j + 1);
        for (int i = 0; i < 32; ++i) {
            printf("%d  ", secret[j][i]);
        }
        printf("\n");
    }

}

int create_keypair(int enable_logging) {
    unsigned char *public_key;
    unsigned char *private_key;
    unsigned char *seed_hf;
    unsigned char *seed;
    unsigned char seed_h[33] = "01234567890123456789012345678901";

    hipMalloc(&public_key, 32 * sizeof(unsigned char));
    hipMalloc(&private_key, 64 * sizeof(unsigned char));
    hipMalloc(&seed, 32 * sizeof(unsigned char));
    hipHostMalloc(&seed_hf, 32 * sizeof(unsigned char));

    // Test whether create seed __host__ function works
    ed25519_kernel_create_seed(seed_hf,1);
    hipMemcpy(seed, seed_hf, 32 * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Use a predefined seed to enbale reproducibility
    hipMemcpy(seed, seed_h, 32 * sizeof(unsigned char), hipMemcpyHostToDevice);

    ed25519_kernel_create_keypair_batch<<<1,1>>>(public_key, private_key, (const unsigned char*) seed, 1);

    if (enable_logging) {
        unsigned char public_key_h[32];
        unsigned char private_key_h[64];

        hipMemcpy(public_key_h, public_key, 32 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        hipMemcpy(private_key_h, private_key, 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);

        //error = hipGetLastError();
        //if (error != hipSuccess) {
        //    printf("Error cuda ed25519 logging: %s \n", hipGetErrorString(error));
        //}

        display_details(public_key_h, private_key_h);
    }

    hipFree(public_key);
    hipFree(private_key);
    hipFree(seed);
    hipHostFree(seed_hf);
    return 1;
}

int main(int argc, char **argv) {
    int enable_logging = 0;
    for (int i = 0 ; i < argc; ++i) {
        if (strcmp(argv[i], "--logging") == 0 || strcmp(argv[i], "-l") == 0) {
            enable_logging = 1;
        }
    }

    printf(((enable_logging)? "\n\n---------- Test batch with single key pair ----------\n\n": ""));
    assert(create_keypair(enable_logging));
    return 0;
}
