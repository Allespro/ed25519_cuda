#include "hip/hip_runtime.h"
#include "keccakf.cuh"
__device__ void keccakf(uint64_t s[25])
{
    int i, j, round;
    uint64_t t, bc[5];

    for(round = 0; round < 24; round++) {

        /* Theta */
        for(i = 0; i < 5; i++)
            bc[i] = s[i] ^ s[i + 5] ^ s[i + 10] ^ s[i + 15] ^ s[i + 20];

        for(i = 0; i < 5; i++) {
            t = bc[(i + 4) % 5] ^ SHA3_ROTL64(bc[(i + 1) % 5], 1);
            for(j = 0; j < 25; j += 5)
                s[j + i] ^= t;
        }

        /* Rho Pi */
        t = s[1];
        for(i = 0; i < 24; i++) {
            j = keccakf_piln[i];
            bc[0] = s[j];
            s[j] = SHA3_ROTL64(t, keccakf_rotc[i]);
            t = bc[0];
        }

        /* Chi */
        for(j = 0; j < 25; j += 5) {
            for(i = 0; i < 5; i++)
                bc[i] = s[j + i];
            for(i = 0; i < 5; i++)
                s[j + i] ^= (~bc[(i + 1) % 5]) & bc[(i + 2) % 5];
        }

        /* Iota */
        s[0] ^= keccakf_rndc[round];
    }
}