#include "keccakf.cuh"


__device__ void onion_address(unsigned char *public_key, uint8_t *checksum) {
    for (int i = 0; i < 32; ++i) checksum[15 + i] = public_key[i];
    checksum[47] = 0x03;
    // pad checksum up r bits
    checksum[48] = 0x01; // pad10*1 start
    checksum[135] = 0x80; // pad10*1 end

    // cast checksum to S for keccakf implementation (blocks of 64 bits)
    uint64_t *S[25];
    for (size_t i = 0; i < 25; i++) {
        S[i] = (uint64_t *)&checksum[i * 8];
    }

    // run the keccakf transformation
    keccakf(*S); 
}